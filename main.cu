#include "hip/hip_runtime.h"
#include "../Utilidades/book.h"

#define N 10

__global__ void add(int *a, int *b, int *c){
    int tidx = blockIdx.x;
    int tidy = blockIdx.y;
    if(tidx < N && tidy < N)
        c[tidx][tidy] = a[tid][tidy] + b[tid][tidy];
}

int main(){
    int a[N][N], b[N][N], c[N][N];
    int *dev_a, *dev_b, *dev_c;

    HANDLE_ERROR(hipMalloc((void**)&dev_a, N*sizeof(int)));
    HANDLE_ERROR(hipMalloc((void**)&dev_b, N*sizeof(int)));
    HANDLE_ERROR(hipMalloc((void**)&dev_c, N*sizeof(int)));

    for(int i = 0; i < N; i++){
        for(int j = 0; i < N; j++){
            a[i][i] = i;
            b[i][j] = i;
        }
    }

    HANDLE_ERROR(hipMalloc(dev_a,a,N*sizeof(int),hipMemcpyDeviceToHost));
    HANDLE_ERROR(hipMalloc(dev_b,b,N*sizeof(int),hipMemcpyDeviceToHost));

    add<<<N*N,1>>>(dev_a, dev_b, dev_c);

    HANDLE_ERROR(hipMalloc(dev_c,c,N*sizeof(int),hipMemcpyDeviceToHost));

    for(int i = 0; i < N; i++){
        for(int j = 0; j < N; j++)
            printf("%d + %d = %d\n", a[i][j], b[i][j], c[i][j]);
    }

    HANDLE_ERROR(hipFree(dev_a));
    HANDLE_ERROR(hipFree(dev_b));
    HANDLE_ERROR(hipFree(dev_c));
}